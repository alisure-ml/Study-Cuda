
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 6400
#define TPB 32  // 核函数中线程块维度大小

__device__ float scale(int i, int n){
    return ((float)i)/(n-1);
}


__device__ float distance(float x1, float x2){
    return sqrt((x2 - x1) * (x2 - x1));
}


__global__ void distanceKernel(float *d_out, float ref, int n){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    float x = scale(i, n);

    d_out[i] = distance(x, ref);
    printf("i = %2d: dist from %f to %f is %f.\n", i, ref, x, d_out[i]);
}


int main(int argc, char **argv){
    float ref = 0.5f;
    float *d_out = 0;

    hipMalloc(&d_out, N * sizeof(float));

    distanceKernel<<<N/TPB, TPB>>>(d_out, ref, N);

    hipFree(d_out);
    return 0;
}
