
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 12800
#define TPB 640  // 核函数中线程块维度大小


__device__ float my_power(float x1){
    return x1 * x1;
}


__global__ void distanceKernel(float *d_out, int len){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    d_out[i] = my_power(i);
    printf("i = %2d: power is %f.\n", i, d_out[i]);
}


int main(int argc, char **argv){
    float *d_out = 0;

    hipMalloc(&d_out, N * sizeof(float));

    distanceKernel<<<N/TPB, TPB>>>(d_out, N);

    hipFree(d_out);
    return 0;
}
